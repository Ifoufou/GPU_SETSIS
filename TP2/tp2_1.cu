#include "hip/hip_runtime.h"
// tp2_1.cu
//
// This program performs a convolution using the CUDA kernel "conv".
// The convolution itself is rather straightforward: we perform it on the 
// sub-region x => [1, width_max-2] and y => [1, height_max-2], thus the
// boundaries of the image (first/last lines and columns) are not processed.
//
// Better case performance (5 executions) (command: nvcc -Xptxas -O[0-3])
// - compiled with -O0: 163 Gpixel/s (4K image, Jetson TX2 v1)
// - compiled with -O3: 175 Gpixel/s (4K image, Jetson TX2 v1)
//
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#undef STB_IMAGE_IMPLEMENTATION
# define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#undef STB_IMAGE_WRITE_IMPLEMENTATION
#include <time.h>

void cuda_error(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

#define cuda_error_check(err) (cuda_error(err, __FILE__, __LINE__))

__device__ int clamp(int x, int a, int b)
{
	return max(a, min(b, x));
}

__global__
void conv(uint8_t* img_in, uint8_t* img_out, int width, int height)
{
	float const kernel[9] = {
		-1.f, 0.f, 1.f,
    -1.f, 0.f, 1.f,
		-1.f, 0.f, 1.f
	};
	
	// Compute row and column numbers of the pixel's image we're targeting.
  // Each thread is translated of one unit, i.e. idx: [0, max-1] => [1, max]
  // to avoid the 0 row/column.
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;

	// Then, we take care of computing everything from 1 to max-2 
	if (row < height-1 && col < width-1)
	{
		// compute pixel of index (row, col)
		float con = img_in[(row-1)*width+col-1] * kernel[0] + 
                img_in[(row-1)*width+col  ] * kernel[1] +
                img_in[(row-1)*width+col+1] * kernel[2] +
                img_in[(row  )*width+col-1] * kernel[3] +
                img_in[(row  )*width+col  ] * kernel[4] +
                img_in[(row  )*width+col+1] * kernel[5] +
                img_in[(row+1)*width+col-1] * kernel[6] +
                img_in[(row+1)*width+col  ] * kernel[7] +
                img_in[(row+1)*width+col+1] * kernel[8];
		img_out[row*width+col] = clamp((int)con, 0, 255);
	}
}

int main(int argc, char* argv[])
{
	char const *const filename = "image.jpg";
	int width = 0, height = 0, nchannels = 0;
	// request to convert image to gray
	int const desired_channels = 1;

	// load the image
	uint8_t *image_data = stbi_load(filename, &width, &height, 
		                              &nchannels, desired_channels);

	// check for errors
	if (!image_data || !width || !height || !nchannels) {
		printf("Error loading image %s\n" , filename);
		return -1;
	}

	int const nb_bytes = width*height*desired_channels*sizeof(uint8_t);
	uint8_t* img_gpu_in  = NULL;
	uint8_t* img_gpu_out = NULL;
	cuda_error_check(hipMalloc((void**)&img_gpu_in, nb_bytes));
	cuda_error_check(hipMalloc((void**)&img_gpu_out, nb_bytes));
	cuda_error_check(hipMemcpy(img_gpu_in, image_data, nb_bytes, hipMemcpyHostToDevice));

	// invoke blocks of size 16x16
	dim3 const blockSize(16, 16, 1);
	dim3 const gridSize(ceil(width/16.0f), ceil(height/16.0f), 1);

	double t_moy = 0.;
	clock_t t;

	for (unsigned i = 0; i < 100; i++) {
		t = clock();
    // invoke blocks of size 16x16 
		conv<<<gridSize, blockSize>>>(img_gpu_in, img_gpu_out, width, height);
		t = clock() - t;
		t_moy = t_moy + t;
  }

	t_moy = t_moy/100.0;
	double const elapsed_time_seconds = ((double)t_moy)/CLOCKS_PER_SEC;
  // measure the performance
	printf("Elapsed time : %lf seconds, pixel/second : %lf\n", elapsed_time_seconds, width*height/elapsed_time_seconds);

	uint8_t * filtered_image = (uint8_t*)malloc(nb_bytes);
  cuda_error_check(hipMemcpy(filtered_image, img_gpu_out, nb_bytes, hipMemcpyDeviceToHost));

	char const *const outfilename = "image_convoluted.png";
	int const stride = width * 1;

	// save the image
	if (!stbi_write_png(outfilename, width, height, 1, filtered_image, stride)) {
		// use the image data
		// release the image memory buffer
    cuda_error_check(hipFree(img_gpu_in));
  	cuda_error_check(hipFree(img_gpu_out));
		free(image_data);
    free(filtered_image);
		return 0;
	}
	return -1;
}

